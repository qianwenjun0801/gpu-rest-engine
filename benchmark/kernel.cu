#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void empty_kernel()
{
}

void kernel_wrapper(hipStream_t stream)
{
    empty_kernel<<<1, 1, 0, stream>>>();
    hipError_t st = hipStreamSynchronize(stream);
    if (st != hipSuccess)
        throw std::invalid_argument("could not launch CUDA kernel");
}
